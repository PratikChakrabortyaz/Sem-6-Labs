#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void computeSine(float *input, float *output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = sin(input[idx]);
    }
}

int main() {
    int N = 1024; 
    float *input, *output;
    float *d_input, *d_output;


    input = new float[N];
    output = new float[N];


    for (int i = 0; i < N; i++) {
        input[i] = (float)i * 0.01f; 
    }


    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, N * sizeof(float));


    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(256, 1, 1); 


    dim3 dimGrid(ceil(N / 256.0), 1, 1); 


    computeSine<<<dimGrid, dimBlock>>>(d_input, d_output, N);


    hipDeviceSynchronize();


    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);


    printf("First 5 sine values:\n");
    for (int i = 0; i < 5; i++) {
        printf("sin(%.2f) = %.6f\n", input[i], output[i]);
    }


    delete[] input;
    delete[] output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
