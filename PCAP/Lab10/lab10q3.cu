#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define TILE_WIDTH 16
#define MAX_MASK_WIDTH 64  

__constant__ int d_mask[MAX_MASK_WIDTH]; 

__global__ void convolution1D_tiled(int *input, int *output, int width, int mask_width) {
    extern __shared__ int N_s[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    int r = mask_width / 2;  
    int shared_idx = tid + r;

    // Load main element
    if (i < width)
        N_s[shared_idx] = input[i];
    else
        N_s[shared_idx] = 0;

    // Load left halo
    if (tid < r) {
        int halo_idx = i - r;
        N_s[tid] = (halo_idx >= 0) ? input[halo_idx] : 0;
    }

    // Load right halo
    if (tid >= blockDim.x - r) {
        int halo_idx = i + r;
        if (halo_idx < width)
            N_s[shared_idx + r] = input[halo_idx];
        else
            N_s[shared_idx + r] = 0;
    }

    __syncthreads();

    // Compute convolution
    int sum = 0;
    if (i < width) {
        for (int j = 0; j < mask_width; j++) {
            sum += d_mask[j] * N_s[shared_idx - r + j];
        }
        output[i] = sum;
    }
}

void printArray(int *array, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", array[i]);
    }
    printf("\n");
}

int main() {
    int width, mask_width;

    printf("Enter the size of the input array: ");
    scanf("%d", &width);
    printf("Enter the size of the mask array (<= %d): ", MAX_MASK_WIDTH);
    scanf("%d", &mask_width);

    if (mask_width > MAX_MASK_WIDTH) {
        printf("Error: mask size exceeds maximum allowed size for constant memory.\n");
        return 1;
    }

    int *h_input, *h_mask, *h_output;
    int *d_input, *d_output;

    size_t bytes_input = width * sizeof(int);
    size_t bytes_mask = mask_width * sizeof(int);

    h_input = (int *)malloc(bytes_input);
    h_mask = (int *)malloc(bytes_mask);
    h_output = (int *)malloc(bytes_input);

    printf("Enter the elements of the input array: ");
    for (int i = 0; i < width; i++) {
        scanf("%d", &h_input[i]);
    }

    printf("Enter the elements of the mask array: ");
    for (int i = 0; i < mask_width; i++) {
        scanf("%d", &h_mask[i]);
    }

    hipMalloc(&d_input, bytes_input);
    hipMalloc(&d_output, bytes_input);

    hipMemcpy(d_input, h_input, bytes_input, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), h_mask, bytes_mask);

    int shared_mem_size = (TILE_WIDTH + mask_width - 1) * sizeof(int);

    dim3 dimGrid((int)ceil((float)width / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH);

    convolution1D_tiled<<<dimGrid, dimBlock, shared_mem_size>>>(d_input, d_output, width, mask_width);

    hipMemcpy(h_output, d_output, bytes_input, hipMemcpyDeviceToHost);

    printf("Output Array:\n");
    printArray(h_output, width);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_mask);
    free(h_output);

    return 0;
}
